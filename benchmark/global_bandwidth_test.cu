#include "hip/hip_runtime.h"

#include "chrono"
#include <cstdlib>
#include <iostream>
using namespace std;
using namespace chrono;

extern "C" __global__ void global_bandwidth_test(double* data, double* cp, int n){
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int range = blockDim.x*gridDim.x;
    for(int i=x;i<n;i+=range){
        cp[i] = data[i];
    }
}

extern "C" __global__ void global_bandwidth_extra_cost_test(double* data, double* cp, int n){
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int range = blockDim.x*gridDim.x;
    for(int i=x;i<n;i+=range){
        
    }
}

int main(){
    // double* h_data = (double*)malloc(sizeof(double)*4096*4096);
    double* d_data, *d_cp;
    hipMalloc(&d_data, sizeof(double)*4096*4096);
    hipMalloc(&d_cp, sizeof(double)*4096*4096);
    hipDeviceSynchronize();
    auto start = system_clock::now();
    global_bandwidth_test<<<4096*4096/64/256, 256>>>(d_data, d_cp, 4096*4096);
    hipDeviceSynchronize();
    // cudaError_t error = cudaGetLastError();
    // printf("%s\n", cudaGetErrorString(error));
    auto end = system_clock::now();
    auto duration = duration_cast<microseconds>(end - start);
    start = system_clock::now();
    global_bandwidth_extra_cost_test<<<4096*4096/128/256, 256>>>(d_data, d_cp, 4096*4096);
    hipDeviceSynchronize();
    end = system_clock::now();
    auto duration_extra_cost = duration_cast<microseconds>(end - start);
    cout<<duration.count()<<endl;
    cout<<duration_extra_cost.count()<<endl;
    double bandwidth = 4096*4096*2*8/(double(duration.count())-double(duration_extra_cost.count()))/1000;
    cout <<  "global bandwidth: " 
     << bandwidth
     << "GB/s" << endl;
}