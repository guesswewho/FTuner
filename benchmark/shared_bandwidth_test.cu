#include "hip/hip_runtime.h"

#include "chrono"
#include <cstdlib>
#include <iostream>
using namespace std;
using namespace chrono;

extern "C" __global__ void shared_bandwidth_test(double* data, double* cp, int n){
    __shared__ double cache[32];
    __shared__ double cache_cp[32];
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int range = blockDim.x*gridDim.x;
    int cache_r = threadIdx.x%32;
    int y;
    for(int i=x;i<n;i+=range){
        cache[cache_r] = data[i];
        for(int j=0;j<1000;j++){
          cache_r = (cache_r+1) % 32;
          cache[cache_r] = cache_cp[cache_r];
          cache_r = (cache_r+1) % 32;
          __syncthreads();
          cache[cache_r] = cache_cp[cache_r];
          // __syncthreads();
          // cache_r = (cache_r+1) % 32;
          // cache[cache_r] = cache_cp[cache_r];
          // __syncthreads();
          // cache_r = (cache_r+1) % 32;
          // cache[cache_r] = cache_cp[cache_r];
        }
        cp[i] = cache[cache_r];
        cp[i] = y;
    }

}

extern "C" __global__ void shared_bandwidth_extra_cost_test(double* data, double* cp, int n){
    __shared__ double cache[32];
    __shared__ double cache_cp[32];
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int range = blockDim.x*gridDim.x;
    int cache_r = threadIdx.x%32;
    int y;
    for(int i=x;i<n;i+=range){
        cache[cache_r] = data[i];
        for(int j=0;j<1000;j++){
          cache_r = (cache_r+1) % 32;
          cache[cache_r] = cache_cp[cache_r];
          cache_r = (cache_r+1) % 32;
          __syncthreads();
          // cache[cache_r] = cache_cp[cache_r];
          // __syncthreads();
          // cache_r = (cache_r+1) % 32;
          // cache[cache_r] = cache_cp[cache_r];
          // __syncthreads();
          // cache_r = (cache_r+1) % 32;
          // cache[cache_r] = cache_cp[cache_r];
        }
        cp[i] = cache[cache_r];
        cp[i] = y;
    }

}

int main(){
    // double* h_data = (double*)malloc(sizeof(double)*4096*4096);
    double* d_data, *d_cp;
    hipMalloc(&d_data, sizeof(double)*4096*4096);
    hipMalloc(&d_cp, sizeof(double)*4096*4096);
    hipDeviceSynchronize();
    auto start = system_clock::now();
    shared_bandwidth_test<<<4096*4096/128/256, 256>>>(d_data, d_cp, 4096*4096);
    hipDeviceSynchronize();
    auto end = system_clock::now();
    auto duration = duration_cast<microseconds>(end - start);
    start = system_clock::now();
    shared_bandwidth_extra_cost_test<<<4096*4096/128/256, 256>>>(d_data, d_cp, 4096*4096);
    hipDeviceSynchronize();
    end = system_clock::now();
    auto duration_extra_cost = duration_cast<microseconds>(end - start);
    cout<<double(duration.count())<<endl;
    cout<<double(duration_extra_cost.count())<<endl;
    double bandwidth = 4096*4096*2*8/(double(duration.count())-double(duration_extra_cost.count()));
    cout <<  "shared memory bandwidth: " 
     << bandwidth
     << "GB/s" << endl;
}